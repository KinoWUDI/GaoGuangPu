#include "hip/hip_runtime.h"

/*
	����˷���CPU�汾��GPU�汾�ĶԱ�
*/

#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <Windows.h>
#include <string>
#include <malloc.h>
#include <omp.h>

// typedef int gpuByte;

//����ָʾ��ͬ��GPU �Ż��汾
enum Type
{
	Mode1 = 1,   //Mode 1 :��ÿһ��C[i][j]���ֱ����һ���߳�
	Mode2 = 2     //Mode 2 :����һ���߳���������һ��C[i][j]��ͨ��C(i,j) = sum { A(i,k)*B(k,j) }���֣����ǻ�������ϸ�Ȼ��֣�
				 //           sub(i,j) = sum{A(i,ksub+offsetA)*B(ksub+offsetB,j)}  0<=ksub < blockSize
				 //            C(i, j) = sum{ Csub(i, j) }
				 //            ���ǰѾ���ֳ�n*n������ӿ飬Ȼ��ÿһ��block��������ӿ�i �� �ӿ�j���ӳ˻���������Ϻ��������ɡ�������Ҫʹ���˹����Դ����Ż���
};

extern "C" hipError_t addWithCuda(double* c, const double* a, const double* b, unsigned int WA, unsigned int HA, unsigned int WB, unsigned int HB, Type mode);


/* gpuMatMultKernel��GPU�¾���˷��˺���
*  a:��һ������ָ�룬��ʾa[M][N]
*  b:�ڶ�������ָ�룬��ʾb[N][S]
*  c:������󣬱�ʾc[M][S]
*/
__global__ void MatrixMulGPU_1(double* c, const double* a, const double* b, unsigned int WA, unsigned int WB)
{
	int sum = 0;
	//�ҳ����߳����ڵ��к���
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	//�߳�Thread(row, col)�������C(row, col)
	for (int i = 0; i < WB; ++i)
	{
		sum += a[row * WA + i] * b[i * WB + col];
	}

	c[row * WB + col] = sum / (512 * 680 - 1);
}

/*
template<int BLOCK_SIZE> __global__ void MatrixMulGPU_2(int* c, const int* a, const int* b, unsigned int WA, unsigned int WB)
{
	// Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	// Index of the first sub-matrix of A processed by the block
	int aBegin = WA * BLOCK_SIZE * by;

	// Index of the last sub-matrix of A processed by the block
	int aEnd = aBegin + WA - 1;

	// Step size used to iterate through the sub-matrices of A
	int aStep = BLOCK_SIZE;

	// Index of the first sub-matrix of B processed by the block
	int bBegin = BLOCK_SIZE * bx;

	// Step size used to iterate through the sub-matrices of B
	int bStep = BLOCK_SIZE * WB;

	// Csub is used to store the element of the block sub-matrix
	// that is computed by the thread
	int Csub = 0;

	// Loop over all the sub-matrices of A and B
	// required to compute the block sub-matrix
	for (int i = aBegin, j = bBegin;
		i <= aEnd;
		i += aStep, j += bStep)
	{

		// Declaration of the shared memory array As used to
		// store the sub-matrix of A
		__shared__ int As[BLOCK_SIZE][BLOCK_SIZE];

		// Declaration of the shared memory array Bs used to
		// store the sub-matrix of B
		__shared__ int Bs[BLOCK_SIZE][BLOCK_SIZE];

		// Load the matrices from device memory
		// to shared memory; each thread loads
		// one element of each matrix
		As[ty][tx] = a[i + WA * ty + tx];
		Bs[ty][tx] = b[j + WB * ty + tx];

		// Synchronize to make sure the matrices are loaded
		__syncthreads();

		// Multiply the two matrices together;
		// each thread computes one element
		// of the block sub-matrix
#pragma unroll

		for (int k = 0; k < BLOCK_SIZE; ++k)
		{
			Csub += As[ty][k] * Bs[k][tx];
		}

		// Synchronize to make sure that the preceding
		// computation is done before loading two new
		// sub-matrices of A and B in the next iteration
		__syncthreads();
	}

	// Write the block sub-matrix to device memory;
	// each thread writes one element
	int k = WB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
	c[k + WB * ty + tx] = Csub;
}
*/

//print the matrix
void printMatrix(int* m_Matrix, int W, int H)
{
	for (int i = 0; i < W * H; ++i)
	{
		printf("%d ", m_Matrix[i]);
		if (i % W == 0 && i != 0) printf("\n");
	}
	printf("\n");
}


extern "C" double* main3(int* datatemp,int dim)
{
	const int width_A = 512 * 680;
	const int height_A = dim;
	const int width_B = dim;
	const int height_B = 512 * 680;

	double* B = (double*)malloc(sizeof(double) * height_B * width_B);
	double* A = (double*)malloc(sizeof(double) * height_A * width_A);
	//int* C = (int*)malloc(sizeof(int) * height_A * width_B);
	double* D = (double*)malloc(sizeof(double) * height_A * width_B);
	//int* E = (int*)malloc(sizeof(int) * height_A * width_B);

	//memset Ϊ��ʼ�� ��0
	//memset(A, 0.0, sizeof(int) * height_A * width_A);
	//memset(B, 0.0, sizeof(int) * height_B * width_B);
	//memset(C, 0.0, sizeof(int) * height_A * width_B);
	memset(D, 0.0, sizeof(double) * height_A * width_B);
	//memset(E, 0.0, sizeof(int) * height_A * width_B);

	//��ȡ����߳���
	int numProcs = omp_get_num_procs();

	// �� A/B���� ��ֵ
	int i;

	for (i = 0; i < height_A * width_A; ++i)
	{
		A[i] = datatemp[i];
		B[i] = datatemp[i];
	}

	//GPU mode1
	Type m_Mode = Mode1;

	unsigned int tick2 = GetTickCount();
	hipError_t cudaStatus = addWithCuda(D, A, B, width_A, height_A, width_B, height_B, m_Mode);

	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "addWithCuda failed!\n");
		//return 1;
	}
	//printf("GPU mode1 use time : %dms\n", GetTickCount() - tick2);


	////GPU mode2
	//m_Mode = Mode2;
	//unsigned int tick3 = GetTickCount();
	//cudaStatus = addWithCuda(E, A, B, width_A, height_A, width_B, height_B, m_Mode);
	//if (cudaStatus != hipSuccess)
	//{
	//    fprintf(stderr, "addWithCuda failed!\n");
	//    return 1;
	//}
	//printf("GPU mode2 use time : %dms\n", GetTickCount() - tick3);

	//���������� �鿴
	//printMatrix(C, 3, 3);

	//Ψһ��Ҫ�����
	//printMatrix(D, 3, 3);

	//printMatrix(E, 3, 3);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceReset failed!");
		//return 1;
	}

	return D;
}

// Helper function for using CUDA to add vectors in parallel.
extern "C" hipError_t addWithCuda(double* c, const double* a, const double* b, unsigned int WA, unsigned int HA, unsigned int WB, unsigned int HB, Type mode)
{
	double* dev_a = 0;
	double* dev_b = 0;
	double* dev_c = 0;
	hipError_t cudaStatus;


	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, HA * WB * sizeof(double)); //HA 224 WB 224
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	cudaStatus = hipMalloc((void**)&dev_a, HA * WA * sizeof(double)); //HA 224 WA 512*680
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	cudaStatus = hipMalloc((void**)&dev_b, HB * WB * sizeof(double)); //HB 512*680 WB 224
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, HA * WA * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
	}

	cudaStatus = hipMemcpy(dev_b, b, HB * WB * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
	}

	//Ϊÿһ��C[i][j]����һ���߳̽��м���
	int block_size = 1;

	dim3 Threads(block_size, block_size);
	dim3 Blocks(WB / block_size, HA / block_size);

	// Launch a kernel on the GPU with one thread for each element.
	if (mode == Mode1)
	{

		MatrixMulGPU_1 << <Blocks, Threads >> > (dev_c, dev_a, dev_b, WA, WB);

	}

	//if (mode == Mode2)
	//{

	//	MatrixMulGPU_2<16> << <Blocks, Threads >> > (dev_c, dev_a, dev_b, WA, WB);

	//}

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, HA * WB * sizeof(double), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
	}

	return cudaStatus;
}

