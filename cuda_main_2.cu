#include <stdio.h>
#include <stdlib.h>
#include <time.h>

//CUDA RunTime API
#include <hip/hip_runtime.h>

#include "hip/device_functions.h"

#define THREAD_NUM 256

#define raw_row 512*680
#define raw_column 224

#define MATRIX_SIZE raw_row*raw_column

const int blocks_num = 16;


//// __global__ ���� ���м������˷�
//__global__ static void matMultCUDA(Imagdata* imagdata, CovData* imagecov, cBYTE* datatemp, clock_t* time)
//{
//
//	//��ʾĿǰ�� thread �ǵڼ��� thread���� 0 ��ʼ���㣩
//	const int tid = threadIdx.x;
//
//	//��ʾĿǰ�� thread ���ڵڼ��� block���� 0 ��ʼ���㣩
//	const int bid = blockIdx.x;
//
//	//// ֻ���÷���
//	////�� bid �� tid �������� thread Ӧ�ü���� row �� column
//	//const int idx = bid * THREAD_NUM + tid;
//	//const int row = idx / n;
//	//const int column = idx % n;
//
//	//��¼���㿪ʼ��ʱ��
//
//	//ֻ�� thread 0���� threadIdx.x = 0 ��ʱ�򣩽��м�¼��ÿ�� block �����¼��ʼʱ�估����ʱ��
//	if (tid == 0) time[bid] = clock();
//	if (tid == 0)
//		printf("Hello thread %d, f=%f\n", tid);
//
//	/*
//	����Э������̿�ʼ
//	*/
//
//	long count;
//	long count2;
//
//	imagecov->row = imagdata->Dim;
//	imagecov->column = imagdata->Dim;
//
//	for (int i = 0; i < imagdata->Dim; ++i) {
//		count = 0;
//		int j = 0;
//		for (j = 0; j < imagdata->imgSize; ++j) {
//			count += imagdata->data[i * imagdata->imgSize + j];
//		}
//		int mean = count / imagdata->imgSize;
//		for (j = 0; j < imagdata->imgSize; ++j) {
//			datatemp[i * imagdata->imgSize + j] = imagdata->data[i * imagdata->imgSize + j] - mean;
//		}
//	}
//	//cout << "(imgcompute.cpp)���ؾ�ֵ��������\n" << endl;
//	printf("(imgcompute.cpp)���ؾ�ֵ��������...\n");
//	//-----------------------------------------------------------------------------------------------------------
//	//�˲�������ʱ��Ϊ45�롣���������Ϊ21�롣��ѭ���μ��룬ʱ������Ϊ12s
//
//	for (int i = 0; i < imagdata->Dim; ++i)
//	{
//		int j;
//		for (j = i; j < imagdata->Dim; ++j)
//		{
//			count2 = 0;
//			int k;
//			//for (k = 0; k < imagdata->imgSize; ++k) {
//
//			//	count2 += datatemp[i * imagdata->imgSize + k] * datatemp[j * imagdata->imgSize + k];
//			//}
//			for (k = 0; k < imagdata->imgSize; ++k) {
//
//				count2 += datatemp[i * imagdata->imgSize + k] * datatemp[j * imagdata->imgSize + k];
//			}
//			imagecov->data[i * (imagecov->row) + j] = count2 / (imagdata->imgSize - 1);
//			if (i != j) {
//				imagecov->data[j * (imagecov->row) + i] = count2 / (imagdata->imgSize - 1);
//			}
//		}
//	}
//
//	delete[]datatemp;
//	printf("(imgcompute.cpp)Э����������\n");
//
//	/*
//	����Э������̽���
//	*/
//
//
//	//����ʱ��,��¼�����ֻ�� thread 0���� threadIdx.x = 0 ��ʱ�򣩽��У�ÿ�� block �����¼��ʼʱ�估����ʱ��
//	if (tid == 0)
//	{
//		time[bid + blocks_num] = clock();
//	}
//
//}
//
//
//__global__ void cudatest() {
//	printf("success!");
//}
//
//
//extern "C" void cuda_juzhen(RawImag * myraw, Imagdata * imagdata, CovData * imagecov)
//{
//	clock_t* time;
//
//	printf("1\n");
//
//	//CPU��GPU�����ڴ� ͬһ���ṹ��Ҫ�ֿ�������ʵ������
//	//CPU�����ڴ�
//	imagecov->data = new covBYTE[sizeof(covBYTE) * imagdata->Dim * imagdata->Dim];
//	cBYTE* datatemp = new cBYTE[sizeof(cBYTE) * imagdata->rawSize]; // rawSize ��512x680x224,8000W
//
//	printf("2\n");
//
//
//	//cudaMalloc ȡ��һ���Կ��ڴ� 
//	//�ṹ������ڴ�Ҫ����ṹ�屾���Լ��ṹ����ÿһ������
//
//
//	Imagdata* cuda_imagdata;
//	CovData* cuda_imagecov;
//	cBYTE* cuda_datatemp;
//
//	cuda_imagdata = new Imagdata[sizeof(Imagdata)]; // ����ṹ�����飨ָ�룩���ڴ�
//	cuda_imagecov = new CovData[sizeof(CovData)];
//	cuda_datatemp = new cBYTE[sizeof(cBYTE)];
//
//
//	//memset(&cuda_imagdata, 0, sizeof(Imagdata));
//	//memset(&cuda_imagecov, 0, sizeof(CovData));
//	//memset(&cuda_datatemp, 0, sizeof(cBYTE));
//
//
//	cuda_imagdata->Col = imagdata->Col;
//	cuda_imagdata->Row = imagdata->Row;
//	cuda_imagdata->Dim = imagdata->Dim;
//	cuda_imagdata->rawSize = imagdata->rawSize;
//	cuda_imagdata->imgSize = imagdata->imgSize;
//
//	cuda_imagecov->row = imagdata->Dim;
//	cuda_imagecov->column = imagdata->Dim;
//
//
//	cuda_imagdata->data = new covBYTE[sizeof(covBYTE) * imagdata->Dim * imagdata->Dim];;
//	cuda_datatemp = new cBYTE[sizeof(cBYTE) * imagdata->rawSize]; // rawSize ��512x680x224,8000W;
//
//	//cudaMalloc((void**)&cuda_imagdata->Row, sizeof(int));
//	//cudaMalloc((void**)&cuda_imagdata->Col, sizeof(int));
//	//cudaMalloc((void**)&cuda_imagdata->Dim, sizeof(int));
//	//cudaMalloc((void**)&cuda_imagdata->rawSize, sizeof(int));
//	//cudaMalloc((void**)&cuda_imagdata->imgSize, sizeof(int));
//
//	//cudaMalloc((void**)&cuda_imagdata, sizeof(Imagdata));
//	cudaMalloc((void**)&cuda_imagdata->data, sizeof(cBYTE) * imagdata->rawSize);
//
//	printf("2.1\n");
//
//	//cudaMalloc((void**)&cuda_imagecov->row, sizeof(int));
//	//cudaMalloc((void**)&cuda_imagecov->column, sizeof(int));
//
//
//	//cudaMalloc((void**)&cuda_imagecov, sizeof(CovData));
//
//	cuda_imagecov->data = new cBYTE[sizeof(cBYTE) * imagdata->rawSize];
//	cudaMalloc((void**)&cuda_imagecov->data, sizeof(covBYTE) * imagdata->Dim * imagdata->Dim);
//
//	printf("2.3\n");
//
//	cudaMalloc((void**)&cuda_datatemp, sizeof(cBYTE) * imagdata->rawSize);
//
//	cudaMalloc((void**)&time, sizeof(clock_t) * blocks_num * 2);
//
//	printf("3\n");
//
//
//	//cudaMemcpy �������ľ����Ƶ��Կ��ڴ���
//	//cudaMemcpyHostToDevice - ���ڴ渴�Ƶ��Կ��ڴ�
//	//cudaMemcpyDeviceToHost - ���Կ��ڴ渴�Ƶ��ڴ�
//
//	//cudaMemcpy(cuda_imagdata, imagdata, sizeof(cBYTE) * imagdata->rawSize, cudaMemcpyHostToDevice);
//	cudaMemcpy(cuda_imagdata->data, imagdata->data, sizeof(cBYTE) * imagdata->rawSize, cudaMemcpyHostToDevice);
//
//	//cudaMemcpy(cuda_imagecov, imagecov, sizeof(covBYTE) * imagdata->rawSize, cudaMemcpyHostToDevice);
//	cudaMemcpy(cuda_imagecov->data, imagecov->data, sizeof(covBYTE) * imagdata->rawSize, cudaMemcpyHostToDevice);
//
//	cudaMemcpy(cuda_datatemp, datatemp, sizeof(cBYTE) * imagdata->rawSize, cudaMemcpyHostToDevice);
//
//	printf("4\n");
//
//	// ��CUDA ��ִ�к��� �﷨����������<<<block ��Ŀ, thread ��Ŀ, shared memory ��С>>>(����...);
//	//matMultCUDA <<< blocks_num, THREAD_NUM>>> (cuda_imag;data, cuda_imagecov, cuda_datatemp, time);
//	cudatest << < 1, 64 >> > ;
//
//	cudaError_t cudaStatus = cudaGetLastError();
//	if (cudaStatus != cudaSuccess)
//	{
//		fprintf(stderr, "addKernel launch failed: %s\n", cudaGetErrorString(cudaStatus));
//	}
//
//
//	printf("5\n");
//
//	/*�ѽ������ʾоƬ���ƻ����ڴ�*/
//
//	clock_t time_use[blocks_num * 2];
//
//	//cudaMemcpy ��������Դ��и��ƻ��ڴ�
//	Imagdata* new_imagdata;
//	CovData* new_imagecov;
//
//	new_imagdata = new Imagdata[sizeof(Imagdata)]; // ����ṹ�����飨ָ�룩���ڴ�
//	new_imagecov = new CovData[sizeof(CovData)];
//
//	//cudaMemcpy(new_imagdata, cuda_imagdata, sizeof(cBYTE) * imagdata->rawSize, cudaMemcpyDeviceToHost);
//	cudaMemcpy(new_imagdata->data, cuda_imagdata->data, sizeof(cBYTE) * imagdata->rawSize, cudaMemcpyDeviceToHost);
//	//cudaMemcpy(new_imagecov, cuda_imagecov, sizeof(covBYTE) * imagdata->rawSize, cudaMemcpyDeviceToHost);
//	cudaMemcpy(new_imagecov->data, cuda_imagecov->data, sizeof(covBYTE) * imagdata->rawSize, cudaMemcpyDeviceToHost);
//
//	cudaMemcpy(&time_use, time, sizeof(clock_t) * blocks_num * 2, cudaMemcpyDeviceToHost);
//
//	printf("6\n");
//
//	//Free
//	cudaFree(cuda_imagdata->data);
//	//cudaFree(cuda_imagdata);
//
//	cudaFree(cuda_imagecov->data);
//	//cudaFree(cuda_imagdata);
//
//	cudaFree(cuda_datatemp);
//	cudaFree(time);
//
//	//��ÿ�� block ����Ŀ�ʼʱ�䣬������Ľ���ʱ�������ȡ��������ʱ��
//	clock_t min_start, max_end;
//
//	min_start = time_use[0];
//
//	max_end = time_use[blocks_num];
//
//	for (int i = 1; i < blocks_num; i++)
//	{
//		if (min_start > time_use[i]) min_start = time_use[i];
//
//		if (max_end < time_use[i + blocks_num]) max_end = time_use[i + blocks_num];
//	}
//
//	//�˺�������ʱ��
//	clock_t final_time = max_end - min_start;
//
//	printf("gputime: %d\n", final_time);
//
//}
